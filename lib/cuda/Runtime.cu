#include <hip/hip_runtime_api.h>

#include "cuda/Runtime.h"
#include "cuda/CudaErrCategory.h"
#include "support/bad_alloc.h"

namespace cuda {

/* getDeviceCount() */
int getDeviceCount() {
  int count;
  checkCall(hipGetDeviceCount(&count), "getDeviceCount");
  return count;
}

/* setDevice() */
void setDevice(int rank) {
  checkCall(hipSetDevice(rank), "setDevice");
}

/* synchronize() */
void synchronize(const char* message) {
#if CUDART_VERSION >= 4000
  hipError_t error = hipDeviceSynchronize();
#else
  hipError_t error = hipDeviceSynchronize();
#endif /* CUDART_VERSION */
  checkCall(error, std::string("synchronize: ") + message);
}

/* malloc() */
void* malloc(const size_t n, const char* message) {
  void* ret = 0;

  hipError_t error = hipMalloc(reinterpret_cast<void**>(&ret), n);

  if (error) {
    throw support::bad_alloc(
        (support::getErrorCategory<CudaErrCategory>().message(error) + 
         message).c_str());
  }
  return ret;
}

/* free() */
void free(void* ptr, const char* message) {
  hipError_t error = hipFree(ptr);
  if (error) {
    throw support::Exception(error, 
                             support::getErrorCategory<CudaErrCategory>(),
                             message);
  }
}

/* memcpy() */
void memcpyH2D(void* dst, const void* src, const size_t count,
                      const char* message) {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
  checkCall(error, std::string("memcpyH2D: ") + message);
}


void memcpyD2H(void* dst, const void* src, const size_t count,
                      const char* message) {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
  checkCall(error, std::string("memcpyD2H: ") + message);
}

void memcpyD2D(void* dst, const void* src, const size_t count,
                      const char* message) {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
  checkCall(error, std::string("memcpyD2D: ") + message);
}

void memcpyH2H(void* dst, const void* src, const size_t count,
                      const char* message) {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyHostToHost);
  checkCall(error, std::string("memcpyH2H: ") + message);
}
} /* namespace cuda */
