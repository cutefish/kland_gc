#include <hip/hip_runtime_api.h>

#include "cuda/Runtime.h"
#include "cuda/CudaErrCategory.h"
#include "support/bad_alloc.h"

namespace cuda {

inline int getDeviceCount() {
  int count;
  checkCall(hipGetDeviceCount(&count), "getDeviceCount");
  return count;
}

inline void synchronize(const char* message) {
#if CUDART_VERSION >= 4000
  hipError_t error = hipDeviceSynchronize();
#else
  hipError_t error = hipDeviceSynchronize();
#endif /* CUDART_VERSION */
  checkCall(error, std::string("synchronize: ") + message);
}

inline void* malloc(const size_t n, const char* message="") {
  void* ret = 0;

  hipError_t error = hipMalloc(reinterpret_cast<void**>(&ret), n);

  if (error) {
    throw support::bad_alloc(
        (getErrorCategory<CudaErrCategory>().message(error) + 
         message).c_str());
}

inline void memcpyH2D(void* dst, const void* src, const size_t count,
                      const char* message="") {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
  checkCall(error, std::string("memcpyH2D: ") + message);
}

inline void memcpyD2H(void* dst, const void* src, const size_t count,
                      const char* message="") {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
  checkCall(error, std::string("memcpyD2H: ") + message);
}

} /* namespace cuda */
