#include <hip/hip_runtime_api.h>

#include "cuda/Runtime.h"
#include "cuda/CudaErrCategory.h"
#include "support/bad_alloc.h"

namespace cuda {

/* getDeviceCount() */
inline int getDeviceCount() {
  int count;
  checkCall(hipGetDeviceCount(&count), "getDeviceCount");
  return count;
}

/* setDevice() */
inline void setDevice(int rank) {
  checkCall(hipSetDevice(rank), "setDevice");
}

/* synchronize() */
inline void synchronize(const char* message) {
#if CUDART_VERSION >= 4000
  hipError_t error = hipDeviceSynchronize();
#else
  hipError_t error = hipDeviceSynchronize();
#endif /* CUDART_VERSION */
  checkCall(error, std::string("synchronize: ") + message);
}

/* malloc() */
inline void* malloc(const size_t n, const char* message) {
  void* ret = 0;

  hipError_t error = hipMalloc(reinterpret_cast<void**>(&ret), n);

  if (error) {
    throw support::bad_alloc(
        (getErrorCategory<CudaErrCategory>().message(error) + 
         message).c_str());
}

/* free() */
inline void free(void* ptr, const char* message) {
  hipError_t error = hipFree(ptr);
  if (error) {
    throw support::Exception(error, 
                             getErrorCategory<CudaErrCategory>(),
                             message);
  }
}

/* memcpy() */
inline void memcpyH2D(void* dst, const void* src, const size_t count,
                      const char* message) {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
  checkCall(error, std::string("memcpyH2D: ") + message);
}


inline void memcpyD2H(void* dst, const void* src, const size_t count,
                      const char* message="") {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
  checkCall(error, std::string("memcpyD2H: ") + message);
}

inline void memcpyD2D(void* dst, const void* src, const size_t count,
                      const char* message="") {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
  checkCall(error, std::string("memcpyD2D: ") + message);
}

inline void memcpyH2H(void* dst, const void* src, const size_t count,
                      const char* message="") {
  hipError_t error = hipMemcpy(dst, src, count, hipMemcpyHostToHost);
  checkCall(error, std::string("memcpyH2H: ") + message);
}
} /* namespace cuda */
